#include "hip/hip_runtime.h"
// Gcode Gen class

#include "kernel.h"
#include <queue>
#include <fstream>

using namespace std;

// NOTE: everything above 50 is considered a white line
ofstream outputFile;

/**
 * Lines to print onto initial file.
 */
void gcode_prolog(void) {
    // G1 means to extrude
    // G0 means to not extrude
    // comments are denoted by a semicolon

    outputFile << "M190 S50.000000" << endl;
    outputFile << "M109 S215.000000" << endl << endl;

    outputFile << "G21            ;metric values" << endl;
    outputFile << "G90            ;absolute positioning" << endl;
    outputFile << "M82            ;set extruder to absolute mode" << endl;
    outputFile << "M107           ;start with the fan off" << endl;
    outputFile << "G28 X0 Y0      ;move X/Y to min endstops" << endl;
    outputFile << "G28 Z0         ;move Z to min endstops" << endl;
    outputFile << "G0 Z15.0 F9000 ;move the platform down 15mm" << endl;
    outputFile << "G92 E0         ;zero the extruded length" << endl;
    outputFile << "G1 F9000       ;Put printing message on LCD screen" << endl;
    outputFile << "M117 DRAWING..." << endl << endl;

    outputFile << ";Layer count: 1" << endl;
    outputFile << ";LAYER:0" << endl;
    outputFile << "M107           ;Turn off the fan" << endl << endl;
    outputFile << ";G1 requires to extrude" << endl;
    outputFile << ";G0 does not require extrusion" << endl << endl;

    // actual gcode goes below;
    // G0 {speed} X{position} Y{position}

}

/**
 * Ending lines to print onto gcode file
 */
void gcode_epilog(double size_h, double size_w) {
    // actual gcode goes below;
    // G0 {speed} X{position} Y{position}

    outputFile << endl;
    outputFile << ";END GCODE" << endl;
    outputFile << "M104 S0        ;extruder heater off" << endl;
    outputFile << "M140 S0        ;heated bed heater off (if you have it)" << endl;
    outputFile << "G91            ;relative positioning" << endl;
    outputFile << "G28 X0 Y0      ;move X/Y to min endstops, so the head is out of the way" << endl;
    outputFile << "M84            ;steppers off" << endl;
    outputFile << "G90            ;absolute positioning" << endl;
    outputFile << ";pixel height: " << size_h << endl;
    outputFile << ";pixel width: " << size_w << endl;
}

/**
 * checks pixels adjacent to the main pixel located at x and y
 * @param image_2d      image in 2d array of format
 * @param image_visited pixel visitation flags for each pixel in 2d format
 * @param x             x position of pixel to check adjacents
 * @param y             y position of pixel to check adjacents
 */
void next_to(int **image_2d, int **image_visited, int x, int y, float size_h, float size_w) {

    int new_x;
    int new_y;
    double pos_x;
    double pos_y;

    //printf("original pixel\n");
    //printf("pixel[%d][%d] = %d\n", x, y, image_2d[x][y]);
    //printf("checking pixels...\n");

    // look at all pixels surrounding the main pixel in question
    for(int col = 0; col < 3; col++) {
        for(int row = 0; row < 3; row++) {
            new_x = x + col - 1;
            new_y = y + row - 1;
            pos_x = new_x * size_w;
            pos_y = new_y * size_h;

            //printf("checking pixel[%d][%d] = %d\n", new_x, new_y, image_visited[new_x][new_y]);
            if(image_2d[new_x][new_y] >= 50 && image_visited[new_x][new_y] == 0) {
                image_visited[new_x][new_y] = 1;
                printf("pixel[%d][%d] = %d\n", new_x, new_y, image_2d[new_x][new_y]);
                outputFile << "G0" << " F1200" << " X" << pos_x << " Y" << pos_y << endl;
                next_to(image_2d, image_visited, new_x, new_y, size_h, size_w);
            }
        }
    }
}

/**
 * processes through all pixels that have not been visited.
 * @param  image  1d vector of image
 * @param  width  [description]
 * @param  height [description]
 * @return        [description]
 */
int gcode(vector<int> image, int width, int height) {

    int **image_2d;
    image_2d = new int *[width];
    int **image_visited;
    image_visited = new int *[width];
    double size_w = (double)190/width;
    double size_h = (double)190/height;

    cout << "width: " << size_w << endl;
    cout << "height: " << size_h << endl;

    gcode_prolog();

    // rebuild the image in 2d format
    // NOTE: we could probably format in 1d but I didnt want to spend much
    // time on this in case somebody had another idea
    for(int i = 0; i < width; i++) {
        image_2d[i] = new int[height];
        image_visited[i] = new int[height];
        for(int j = 0; j < height; j++) {
            image_2d[i][j] = image[i + (j * (width))];
            image_visited[i][j] = 0;
        }
    }

    // image_2d will have a normal color array where anything above
    // a 50 is considered a path.

    // image_visited will have a 0 if the item has not been visited
    // and a 1 if the image has been visited.

     // iterate through the 2d array(s)
     for(int x = 1; x < (width - 1); x++) {
         for(int y = 1; y < (height - 1); y++) {
             // if the image is grey/white and has not been visited
             if(image_2d[x][y] >= 50 && image_visited[x][y] == 0) {
                //printf("pixel[%d][%d] = %d\n", x, y, image_2d[x][y]);
                outputFile << "G0 F1200 Z0.03\t\t\t;move pencil down" << endl;

                // recursive call once a grey/white pixel has been found
                // and follow up with any pixels which are grey/white
                // immediately next to that
                next_to(image_2d, image_visited, x, y, size_h, size_w);

                outputFile << "G0 F1200 Z15.0\t\t\t;move pencil up" << endl;
             }
         }
     }

     gcode_epilog(size_h, size_w);

    return 0;
}

/**
 * gcode generator wrapper function.
 * @param img         1d vector of image pixel contents
 * @param width       the width of the image
 * @param height      the height of the image
 * @param output_name output of name to be written
 */
void g_gen(vector<int> img, int width, int height, string output_name) {

    // add gcode file extension
    output_name.append(".gcode");

    // open file
    // NOTE: we should probably throw in a try-catch and figure out write flags
    outputFile.open(output_name);

    // call the function
    gcode(img, width, height);

    // close file
    //outputFile.close(output_name);

}
