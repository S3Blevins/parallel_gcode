#include "hip/hip_runtime.h"
// Gcode Gen class

#include "kernel.h"
#include <queue>
#include <fstream>

using namespace std;

// NOTE: everything above 50 is considered a white line
ofstream outputFile;

/**
 * Lines to print onto initial file.
 */
void gcode_prolog(void) {
    // G1 means to extrude
    // G0 means to not extrude
    // comments are denoted by a semicolon

    outputFile << "G21            ;metric values" << endl;
    outputFile << "G90            ;absolute positioning" << endl;
    outputFile << "M82            ;set extruder to absolute mode" << endl;
    outputFile << "M107           ;start with the fan off" << endl;
    outputFile << "G28 X0 Y0      ;move X/Y to min endstops" << endl;
    outputFile << "G28 Z0         ;move Z to min endstops" << endl;
    outputFile << "G0 Z5.0 F9000  ;move the platform down 15mm" << endl;
    outputFile << "G92 E0         ;zero the extruded length" << endl;
    outputFile << "G1 F9000       ;Put printing message on LCD screen" << endl;
    outputFile << "M117 DRAWING..." << endl << endl;

    outputFile << ";Layer count: 1" << endl;
    outputFile << ";LAYER:0" << endl;
    outputFile << "M107           ;Turn off the fan" << endl << endl;
    outputFile << ";G1 requires to extrude" << endl;
    outputFile << ";G0 does not require extrusion" << endl << endl;

    // actual gcode goes below;
    // G0 {speed} X{position} Y{position}

}

/**
 * Ending lines to print onto gcode file
 */
void gcode_epilog() {
    // actual gcode goes below;
    // G0 {speed} X{position} Y{position}

    outputFile << endl;
    outputFile << ";END GCODE" << endl;
    outputFile << "M104 S0        ;extruder heater off" << endl;
    outputFile << "M140 S0        ;heated bed heater off (if you have it)" << endl;
    outputFile << "G91            ;relative positioning" << endl;
    outputFile << "G28 X0 Y0      ;move X/Y to min endstops, so the head is out of the way" << endl;
    outputFile << "M84            ;steppers off" << endl;
    outputFile << "G90            ;absolute positioning" << endl;
}

/**
 * non-recursively checks pixels adjacent to the main pixel located at x
 * and y
 * @param image_2d      image in 2d array of format
 * @param image_visited pixel visitation flags for each pixel in 2d format
 * @param x             x position of pixel to check adjacents
 * @param y             y position of pixel to check adjacents
 * @param height        the pixel height of the image
 * @param width         the pixel width of the image
 */
bool next(int **image_2d, int **image_visited, int x, int y, int height, int width) {
    double pos_x;
    double pos_y;

    int old_x = x;
    int old_y = y;

    int new_x = x;
    int new_y = y;
    vector<int> saved_x;            // saved indices to simulate
    vector<int> saved_y;            // recursion
    bool up = false;
    double size = ((double)180/MAX(width,height));

    // insert the first indices in the stack
    saved_x.push_back(old_x);
    saved_y.push_back(old_y);

    // keep checking the surrounding elements as long as
    // the stack is not empty
    while(!saved_x.empty() && !saved_y.empty()){

        for(int col = 0; col < 3; col++) {
            for(int row = 0; row < 3; row++){
                new_x = old_x + col - 1;
                new_y = old_y + row - 1;

                if (new_x >= width || new_y >= height) {
                    //cout << "going out of bounds\theigth: " << height << "\twidth: " << width << endl;
                    continue;
                }

                pos_x = new_x * size;
                pos_y = new_y * size;

                if (image_2d[new_x][new_y] >= 25 && image_visited[new_x][new_y] == 0) {
                    image_visited[new_x][new_y] = 1;
                    old_x = new_x;
                    old_y = new_y;

                    saved_x.push_back(new_x);
                    saved_y.push_back(new_y);

                    col = 0;
                    row = 0;

                    outputFile << "G0" << " F8000" << " X" << pos_x << " Y" << pos_y << " Z0.03\t\t;pen down"<< endl;

                    break;
                }
            }
        }

        saved_x.pop_back();
        saved_y.pop_back();
        old_x = saved_x.back();
        old_y = saved_y.back();
    }
    return up;
}

/**
 * processes through all pixels that have not been visited.
 * @param  image  1d vector of image
 * @param  width  [description]
 * @param  height [description]
 * @return        [description]
 */
int gcode(vector<int> image, int width, int height) {

    int **image_2d;
    image_2d = new int *[width];
    int **image_visited;
    image_visited = new int *[width];
    double pos_x, pos_y;
    bool up = true;
    double size = ((double)180/MAX(width,height));

    gcode_prolog();

    // rebuild the image in 2d format
    // NOTE: we could probably format in 1d but I didnt want to spend much
    // time on this in case somebody had another idea
    for(int i = 0; i < width; i++) {
        image_2d[i] = new int[height];
        image_visited[i] = new int[height];
        for(int j = 0; j < height; j++) {
            image_2d[i][j] = image[i + (j * (width))];
            image_visited[i][j] = 0;
        }
    }

    // image_2d will have a normal color array where anything above
    // a 50 is considered a path.

    // image_visited will have a 0 if the item has not been visited
    // and a 1 if the image has been visited.

     // iterate through the 2d array(s)
     for(int x = 1; x < (width - 1); x++) {
         for(int y = 1; y < (height - 1); y++) {
             // if the image is grey/white and has not been visited
             if(image_2d[x][y] >= 25 && image_visited[x][y] == 0) {
                //printf("pixel[%d][%d] = %d\n", x, y, image_2d[x][y]);

                // recursive call once a grey/white pixel has been found
                // and follow up with any pixels which are grey/white
                // immediately next to that
                pos_x = x * size;
                pos_y = y * size;

                outputFile << "G0 F8000 X" << pos_x <<  " Y" << pos_y << endl;

                if (up) {
                    outputFile << "G0 F10000 Z0.03\t\t\t;moving down" << endl;
                    up = false;
                }

                up = next(image_2d, image_visited, x, y, height, width);

                if (!up) {
                    outputFile << "G0 F10000 Z3.0\t\t\t;move pencil up" << endl;
                }
             }
         }
     }
     gcode_epilog();

    return 0;
}

/**
 * gcode generator wrapper function.
 * @param img         1d vector of image pixel contents
 * @param width       the width of the image
 * @param height      the height of the image
 * @param output_name output of name to be written
 */
void g_gen(vector<int> img, int width, int height, string output_name) {

    // add gcode file extension
    output_name.append(".gcode");

    // open file
    // NOTE: we should probably throw in a try-catch and figure out write flags
    outputFile.open(output_name);

    // call the function
    gcode(img, width, height);

    // close file
    //outputFile.close(output_name);

}
