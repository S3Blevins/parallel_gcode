#include "hip/hip_runtime.h"
//#include <hip/hip_runtime.h>
#include "kernel.h"
#include "gcode_gen.cu"
#include <stdio.h>

using namespace cimg_library;
using namespace std;

//-------------------------FILTERS FOR DETECTORS--------------------------------
//                      (PREDEFINED AT COMPILE TIME)
//-------------------------Sobel Edge Detector----------------------------------
// sobelFilter x matrix
int Gx_matrix[9] = {1, 0, -1,
                    2, 0, -2,
                    1, 0, -1};

// sobelFilter y matrix
int Gy_matrix[9] = {1, 2, 1,
                    0, 0, 0,
                   -1, -2, -1};

//------------------------Roberts Cross Edge Detector---------------------------
int RGx_matrix[4] = {1, 0,
                     0, -1};
int RGy_matrix[4] = {0, 1,
                    -1, 0};

<<<<<<< HEAD
//------------------------------------------------------------------------------
=======
//------------------------Prewitt Edge Detector---------------------------------
int PGx_matrix[9] = {-1, 0, 1,
                     -1, 0, 1,
                     -1, 0, 1};
int PGy_matrix[9] = {-1, -1, -1,
                      0, 0, 0,
                      1, 1, 1};
>>>>>>> aebdb45a988c7b93f88fd778f312bbc05ab1c85a


__global__
void sobelFilterKernel(int *imageRGB, int *output, int width, int height, int *Gx_array, int *Gy_array, int threshold) {
    int Gx, Gy;
    int length;
    int normalized_pixel;

    //calculate thread locations (threadIDx)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int x = i % width;  // x is where in the matrix x direction.
    int y = (i / width); // y is where in the matrix in the y direction.

    // initialize Gx and Gy intensities to 0 for every pixel
    Gx = 0;
    Gy = 0;
    int RGB;

    if((i < (width * height)) && (x > 0) && (y > 0)  && (x < width - 1) && (y < height - 1)) {

        for(int filter_pos = 0; filter_pos < 9; filter_pos++) {
            int col = filter_pos % 3;
            int row = filter_pos / 3;

            RGB = imageRGB[(x + col - 1) + (width * (y + row - 1))];

            // summation of Gx and Gy intensities
            Gx += Gx_array[filter_pos] * RGB;
            Gy += Gy_array[filter_pos] * RGB;
        }

        // absolute value
        if(Gx < 0) {
            Gx *= -1;
        }
        if(Gy < 0) {
            Gy *= -1;
        }

        // absolute value of intensities
        length = Gx + Gy;

        // normalize the gradient with threshold value (DEFAULT: 2048)
        normalized_pixel = length * 255 / threshold;
        __syncthreads();
        output[x + (width * y)] = normalized_pixel;
    }
     __syncthreads();
}

__global__
void robertFilterKernel(int *imageRGB, int *output, int width, int height, int *RGx_array, int *RGy_array, int threshold) {
    int Gx, Gy;
    int length;
    int normalized_pixel;

    //calculate thread locations (threadIDx)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int x = i % width;
    int y = i / width;

    // loop through pixels x and y
    //for(int x = 1; x < width; x++) {
    //    for(int y = 1; y < height; y++) {

    // initialize Gx and Gy intensities to 0 for every pixel
    Gx = 0;
    Gy = 0;
    int RGB;

    if((i < (width * height)) && (x > 0) && (y > 0)  && (x < width - 1) && (y < height - 1)) {

        for(int filter_pos = 0; filter_pos < 4; filter_pos++) {
            int col = filter_pos % 2;
            int row = filter_pos / 2;

            RGB = imageRGB[(x + col - 1) + (width * (y + row - 1))];

            // summation of Gx and Gy intensities
            Gx += RGx_array[filter_pos] * RGB;
            Gy += RGy_array[filter_pos] * RGB;
        }

        // absolute value
        if(Gx < 0) {
            Gx *= -1;
        }
        if(Gy < 0) {
            Gy *= -1;
        }

        // absolute value of intensities
        length = Gx + Gy;

        // normalize the gradient with threshold value (DEFAULT: 2048)
        normalized_pixel = length * 255 / threshold;
        __syncthreads();
        output[x + (width * y)] = normalized_pixel;
    }
     __syncthreads();
}

/**
 * function will error check the cuda malloc.
 * If error, system will output the location.
 * @params: err which is if the cuda malloc worked or not.
 */
void error_check(hipError_t err) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}

/**
 * edge detection wrapper processes the appropriate flags
 * @param flags       flags to execute
 * @param input_name  image name
 * @param output_name output name for gcode output
 * @param threshold   threshold used for normalizing the sobel filter
 */
void edge_detection_wrapper(char flags, string input_name, string output_name, int threshold, int filter) {
    cimg::exception_mode(0); // silence library exceptions so we can use our own
    CImg<unsigned int> img;

    // dimensions to be set by set when cast by reference below
    int width;
    int height;

    double scaler = 1;

    // open image
    try {
        img.assign(input_name.c_str());
        img.mirror('x');

        width = img.width();
        height = img.height();

    } catch (CImgIOException) {
        cout << "Image file has not been located. Please use an appropriate image." << endl;
        exit(0);
    }

    // scaling
    if(MAX(width,height) > 1024) {
        scaler = ((double)1024/MAX(width,height));

    } else if (MIN(width,height) < 512) {
        scaler = ((double)512/MIN(width,height));

    } else {
        scaler = 1;
    }

    // resize
    img.resize((int)(width * scaler), (int)(height * scaler), 1, 1, 3);

    // reset width
    width = img.width();
    height = img.height();

    // vectorize the image
    vector<int> image_vector = vectorize_img(img);

    // print out some minor metadata (need to finish)
    if(flags & 0x8) {
        metadata(img, threshold);
    }

    // if GPU or CPU processed, call the appropriate function
    if(flags & 0x1) {
        //printf("GPU Processed\n");

        // overwrite the image vector with sobel filter
        image_vector = edge_detection_gpu(image_vector, width, height, threshold, filter);
    } else {
        //printf("CPU Processed\n");

        // overwrite the image vector with filter choice
        image_vector = edge_detection_cpu(image_vector, width, height, threshold, filter);
    }

    // call g-code generator here
    // needs to be linked in makefile
    g_gen(image_vector, width, height, output_name);

    // display the image when the filter has been applied
    if(flags & 0x2 || flags & 0x10) {
        display_img(image_vector, width, height, (flags), output_name);
    }
}

/**
 * image is turned into a vector (un-avoidable overhead)
 * @param  img    image to be converted into a 1D vector
 * @param  width  width is created via pass by reference
 * @param  height height is created via pass by reference
 * @return        vectorized image in terms of integers
 */
vector<int> vectorize_img(CImg<unsigned char> img) {
    int width = img.width();
    int height = img.height();

    vector<int> image_vector(width * height);

    // loop through pixels x and y
    for(int x = 0; x < width; x++) {
        for(int y = 0; y < height; y++) {
           image_vector[x + (y * (width))] = img.atXY(x,y);
           //printf("%i\n", image_vector[x + (y * (*width))]);
        }
    }

    return image_vector;
}

vector<int> edge_detection_gpu(vector<int> img, int width, int height, int threshold, int filter) {
    // convert vector into standard image array
    int* img_array = &img[0];
    int image_size, image_array_size, matrix_array_size;

    // Device IMG_array, device Sobel Filter x, device Sobel Filter y
    int *inputIMG_array, *outputIMG_array, *filterx, *filtery;

    if (filter == 1) {
        image_size = width * height;
        image_array_size = image_size * sizeof(int);
        matrix_array_size = 9 * sizeof(int);
    } else if (filter == 2) {
        image_size = width * height;
        image_array_size = image_size * sizeof(int);
        matrix_array_size = 4 * sizeof(int);
    } else if (filter == 3) {
        printf("Launching Prewitt Edge Detector\n");
        exit(0);
    } else if (filter == 4 ) {
        printf("Launching Frie Chen Edge Detector\n");
        exit(0);
    } else {
        printf("Not a valid filter choice. Please try again.\n");
        exit(1);
    }

    // allocating memory for device variables
    //--------------------------------------------------------------------------
    hipError_t err = hipMalloc((void **) &outputIMG_array, image_array_size);
    error_check(err);

    err = hipMalloc((void **) &inputIMG_array, image_array_size);
    error_check(err);

    err = hipMalloc((void **) &filterx, matrix_array_size);
    error_check(err);

    err = hipMalloc((void **) &filtery, matrix_array_size);
    error_check(err);
    //--------------------------------------------------------------------------

    // Copy array to device memory
    hipMemcpy(inputIMG_array, img_array, image_array_size, hipMemcpyHostToDevice);

    //For timing purposes.
    float ms = 0;
    hipEvent_t begin, end;

    if (filter == 1) {
        // Copy array to device memory
        hipMemcpy(filterx, Gx_matrix, matrix_array_size, hipMemcpyHostToDevice);
        // Copy array to device memory
        hipMemcpy(filtery, Gy_matrix, matrix_array_size, hipMemcpyHostToDevice);
        // Launch kernel (UNSURE OF BLOCKS PER GRID vs THREADS PER BLOCK)
        printf("Launching Sobel Edge Detector\n");
        //timing prep
        hipEventCreate(&begin);
        hipEventCreate(&end);

        //timing start
        hipEventRecord(begin);
        sobelFilterKernel <<< ceil(image_size/256.0), 256 >>> (inputIMG_array, outputIMG_array, width, height, filterx, filtery, threshold);
        //timing stop
        hipEventRecord(end);
    } else if (filter == 2) {
        // Copy array to device memory
        hipMemcpy(filterx, RGx_matrix, matrix_array_size, hipMemcpyHostToDevice);
        // Copy array to device memory
        hipMemcpy(filtery, RGy_matrix, matrix_array_size, hipMemcpyHostToDevice);
        printf("Launching Robert's Edge Detector\n");
        //timing prep
        hipEventCreate(&begin);
        hipEventCreate(&end);

        //timing start
        hipEventRecord(begin);
        robertFilterKernel <<< ceil(image_size/256.0), 256 >>> (inputIMG_array, outputIMG_array, width, height, filterx, filtery, threshold);
        //timing stop
        hipEventRecord(end);
    } else if (filter == 3) {
        printf("Launching Prewitt Edge Detector\n");
<<<<<<< HEAD
=======
        //timing prep
        hipEventCreate(&begin);
        hipEventCreate(&end);

        //timing start
        hipEventRecord(begin);
        prewittFilterKernel <<< ceil(image_size/256.0), 256 >>> (inputIMG_array, outputIMG_array, width, height, filterx, filtery, threshold);
        //timing stop
        hipEventRecord(end);
>>>>>>> aebdb45a988c7b93f88fd778f312bbc05ab1c85a
    } else {
        printf("Not a valid filter, exiting...\n");
        exit(1);
    }
    hipDeviceSynchronize();

    // Start allocating memory for new device variables
    int *filterImageOutput;
    //262144

    // filterPictureOutput is the RGB values of the image normalized to Filter
    filterImageOutput = (int *) malloc(image_array_size);
    if (filterImageOutput == NULL) {
        printf("Could not allocate memory for sobelPictureOutput: failed\n");
        exit(1);
    }

    // Success - This point should have the picture in an output array
    hipMemcpy(filterImageOutput, outputIMG_array, image_array_size, hipMemcpyDeviceToHost);

    hipEventSynchronize(end);
    ms = 0;
    hipEventElapsedTime(&ms, begin, end);

    printf("Time elapsed: %f ms \n", ms);

    // I think this is how we convert an array into a vector?
    vector<int> out(filterImageOutput, filterImageOutput + image_size);
    return out;
}

/**
 * Sobel Edge Detection filter run via the CPU.
 * @param  width     width of the image
 * @param  height    height of the image
 * @param  threshold threshold for normalization for filter
 * @return           vectorized image with filter applied
 */
vector<int> edge_detection_cpu(vector<int> img, int width, int height, int threshold, int filter) {

    int Gx;
    int Gy;

    int col;
    int row;

    int RGB;
    int length;
    int normalized_pixel;

    vector<int> image_vector(width * height);

    // loop through pixels x and y
    for(int x = 1; x < width; x++) {
        for(int y = 1; y < height; y++) {
            // initialize Gx and Gy intensities to 0 for every pixel
            Gx = 0;
            Gy = 0;

            for(int i = 0; i < 9; i++) {
                col = i % 3;
                row = i / 3;

                RGB = img[(x + col - 1) + (width * (y + row - 1))];

                // summation of Gx and Gy intensities
                Gx += Gx_matrix[i] * RGB;
                Gy += Gy_matrix[i] * RGB;
            }

            // absolute value of intensities
            length = abs(Gx) + abs(Gy);

            // normalize the gradient with threshold value (DEFAULT: 2048)
            normalized_pixel = length * 255 / threshold;

            // set pixel value
            image_vector[x + (width * y)] = normalized_pixel;
        }
    }

    return image_vector;
}

/**
 * Used when the flag to view the image is used
 * @param  img        vectorized image
 * @param  width      width of image
 * @param  height     height of image
 * @param  write_flag flag for writing new image to a file
 * @param  output     output name
 * @return            0
 */
int display_img(vector<int> img, int width, int height, int flags, string output) {

    CImg<unsigned char> new_img;
    new_img.assign(width, height, 1, 3);

    for(int x = 0; x <= width; x++) {
        for(int y = 0; y <= height; y++) {
            // Red, Green, and Blue values are all the same
            new_img.atXY(x,y,0) = img[x + (y * width)];
            new_img.atXY(x,y,1) = img[x + (y * width)];
            new_img.atXY(x,y,2) = img[x + (y * width)];
            if(new_img.atXY(x,y) >= 50) {
                //printf("new_img.atXY[%d][%d] = %d\n", x, y, new_img.atXY(x,y));
            }
        }
    }

    new_img.mirror('x');

    // Display the image
    if (flags & 0x2) {
        CImgDisplay main_disp(new_img,"Image");
        while (!main_disp.is_closed()) {
            main_disp.wait();
        }
    }

    // if write_flag exists, then save the image
    if(flags & 0x10) {
        output.append(".bmp");

        new_img.save(output.c_str());
    }

    return 0;
}

/**
 * metadata of image
 * @param  img image object
 * @return     0
 */
int metadata(CImg<unsigned char> img, int threshold) {

    // TODO: add more useful data

    // populate dimensions
    int width = img.width();
    int height = img.height();

    // print out dimensions, etc
    printf("Width: %i\n", width);
    printf("Height: %i\n", height);
    printf("Total Pixel Count: %.2f MP\n", ((float)(width * height))/1000000);
    printf("Threshold value: %d\n", threshold);

    return 0;

}
